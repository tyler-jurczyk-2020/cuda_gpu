#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>

#define BLOCK_SIZE 32

__global__ void conv_forward_kernel(__half *output, const __half *input, const __half *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K, const int out_grid_width)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.
    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int nth_in = blockIdx.z;
    int map = blockIdx.y;
    int w0 = threadIdx.x;
    int h0 = threadIdx.y;
    int h_base = (blockIdx.x / out_grid_width) * BLOCK_SIZE;
    int w_base = (blockIdx.x % out_grid_width) * BLOCK_SIZE;
    int h = h_base + h0;
    int w = w_base + w0;
    __half sum = 0;

    for(int c = 0; c < Channel; c++) {
        for(int p = 0; p < K; p++) {
            for(int q = 0; q < K; q++) {
                if(h < Height_out && w < Width_out) {
                    sum = __hadd(sum, __hmul(in_4d(nth_in, c, h + p, w + q), mask_4d(map, c, p , q)));
                }
            }
        }
    }
    if(h < Height_out && w < Width_out) {
        out_4d(nth_in, map, h, w) = (float) sum;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    *device_input_ptr = (float *) host_input;
    *device_mask_ptr = (float *) host_mask;
    *device_output_ptr = (float *) host_output;
    /*
    hipMalloc(device_input_ptr, input_size);
    hipMalloc(device_mask_ptr, mask_size);

    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);
    */
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}

__host__ void convert2Half(__half *output, const float *input, int nmem) {
    for(int i = 0; i < nmem; i++) {
        output[i] = __float2half(input[i]);
    }
}

__host__ void convert2Float(float *output, const __half *input, int nmem) {
    for(int i = 0; i < nmem; i++) {
        output[i] = __half2float(input[i]);
    }
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    __half *half_device_output;
    __half *half_device_input;
    __half *half_device_mask;

    size_t half_input_size = Batch * Channel * Width * Height * sizeof(__half);
    size_t half_output_size = Batch * Map_out * (Width - K + 1) * (Height - K + 1) * sizeof(__half);
    size_t half_mask_size = Map_out * Channel * K * K * sizeof(__half); 
    hipMalloc(&half_device_input, half_input_size);
    hipMalloc(&half_device_output, half_output_size);
    hipMalloc(&half_device_mask, half_mask_size);

    __half *temp_input;
    __half *temp_mask;
    __half *temp_output;
    temp_input = (half *)malloc(half_input_size);
    temp_mask = (half *)malloc(half_mask_size);
    convert2Half(temp_input, device_input, Batch * Channel * Width * Height);
    convert2Half(temp_mask, device_mask, Map_out * Channel * K * K);

    hipMemcpy(half_device_input, temp_input, half_input_size, hipMemcpyHostToDevice);
    hipMemcpy(half_device_mask, temp_mask, half_mask_size, hipMemcpyHostToDevice);

    // Set the kernel dimensions and call the kernel
    int output_width = Width - K + 1;
    int output_height = Height - K + 1;
    int output_width_tiles = ceil(1.0f*output_width/BLOCK_SIZE);
    int output_height_tiles = ceil(1.0f*output_height/BLOCK_SIZE);

    // Using lecture slide implementation
    dim3 grid_dim(output_width_tiles * output_height_tiles, Map_out, Batch);
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);

    conv_forward_kernel<<<grid_dim, block_dim>>>(half_device_output, half_device_input, half_device_mask, Batch, Map_out, Channel, Height, Width, K, output_width_tiles);

    hipDeviceSynchronize();

    temp_output = (__half *)malloc(half_output_size);
    hipMemcpy(temp_output, half_device_output, half_output_size, hipMemcpyDeviceToHost);
    
    convert2Float(device_output, temp_output, Batch * Map_out * (Width - K + 1) * (Height - K + 1));

    hipFree(half_device_input);
    hipFree(half_device_mask);
    free(temp_input);
    free(temp_mask);
    free(temp_output);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Free device memory
    free(device_input);
    hipFree(device_output);
    free(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
