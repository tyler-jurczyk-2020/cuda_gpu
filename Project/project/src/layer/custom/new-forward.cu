#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define BLOCK_SIZE 32

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    size_t input_size = Batch * Channel * Width * Height * sizeof(float);
    size_t output_size = Batch * Map_out * (Width - K + 1) * (Height - K + 1) * sizeof(float);
    size_t mask_size = K * K * sizeof(float); 
    hipMalloc(device_input_ptr, input_size);
    hipMalloc(device_output_ptr, output_size);
    hipMalloc(device_mask_ptr, mask_size);

    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    conv_forward_gpu(*device_output_ptr, *device_input_ptr, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);

    conv_forward_gpu_epilog((float *)host_output, *device_output_ptr, *device_input_ptr, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int output_width = Width - K + 1;
    int output_height = Height - K + 1;
    int output_width_tiles = ceil(1.0f*output_width/BLOCK_SIZE);
    int output_height_tiles = ceil(1.0f*output_height/BLOCK_SIZE);

    // May need to adjust grid dimensions later on
    dim3 grid_dim(Map_out, output_width_tiles * output_height_tiles, Batch);
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);

    conv_forward_kernel<<<grid_dim, block_dim>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    size_t output_size = Batch * Map_out * (Width - K + 1) * (Height - K + 1) * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
