#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < len) {
    out[i] = in1[i] + in2[i];
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  size_t size = inputLength * sizeof(float);

  //@@ Allocate GPU memory here
  
  hipMalloc(&deviceInput1, size);
  hipMalloc(&deviceInput2, size);
  hipMalloc(&deviceOutput, size);

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 gridDim(ceil(inputLength/256.0f), 1, 1);
  dim3 blockDim(256, 1, 1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
